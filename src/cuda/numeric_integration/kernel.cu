#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include "kernel.h"

// cuda kernel function always return void, pass in pointer to write
__global__ void kernel_integrate(
    float *sumArrayW, int numSteps, int numTerms, float stepSize)
{
    auto step = blockIdx.x * blockDim.x + threadIdx.x;
    if (step < numSteps)
    {
        int x = stepSize * step;
        sumArrayW[x] = sinAtX(x, numTerms);
    }
}

void run_kernel_integrate(int blocks, int threadsInBlock, int numSteps, int numTerms, float stepSize)
{
    // gpu vector can only be compiled in cuda
    thrust::device_vector<float> sumArrayW(numSteps);
    float *ptr = thrust::raw_pointer_cast(&sumArrayW[0]);

    kernel_integrate<<<blocks, threadsInBlock>>>(
        ptr,
        numSteps,
        numTerms,
        stepSize);
    return;
}
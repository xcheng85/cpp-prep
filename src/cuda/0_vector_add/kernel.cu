#include "hip/hip_runtime.h"
#include <algorithm>
#include <assert.h>
#include <thread>
#include <chrono>
#include <thrust/device_vector.h>
#include <cuPredefines.h>
#include "kernel.h"

using namespace std::chrono;

__constant__ float constScale[NUM_SCALEFACTOR];

void setScaleFactor(const float *coeff)
{
    hipMemcpyToSymbol(HIP_SYMBOL(constScale), coeff, NUM_SCALEFACTOR * sizeof(float));
}

// v1: to be overwritten
__global__ void vectorAdd(float *v1, float *v2, int N)
{
    extern __shared__ float intermediates[];
    // only supports 1d
    int globalThreadId = blockDim.x * blockIdx.x + threadIdx.x;
    // two global memory access
    // debug in kernel
    if (globalThreadId < N)
    {
        printf("calling kernel: %d\n", globalThreadId);
        v1[globalThreadId] += v2[globalThreadId];
        v1[globalThreadId] *= constScale[0];
    }
}

thrust::host_vector<float> runVectorAdd(thrust::host_vector<float> &v1,
                                        thrust::host_vector<float> &v2,
                                        dim3 numThreadsInBlock)
{
    thrust::host_vector<float> res;
    assert(v1.size() == v2.size());
    auto N = v1.size();
    thrust::device_vector<float> v1Device(N), v2Device(N);

    // copy data from host to device through PCI-bus
    // deep down using hipMemcpy

    v1Device = v1;
    v2Device = v2;

    float *ptr1 = thrust::raw_pointer_cast(&v1Device[0]);
    float *ptr2 = thrust::raw_pointer_cast(&v2Device[0]);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    auto err = hipGetLastError();
    if (err != hipSuccess)
    {
        std::cerr << hipGetErrorString(err) << std::endl;
    }
    hipEventCreate(&stop);
    // stream 0
    hipEventRecord(start, 0);
    dim3 numBlocksPerGrid((N + numThreadsInBlock.x - 1) / numThreadsInBlock.x);

    // 2d
    // dim3 numBlocksPerGrid(32, 32, 1);
    int blockSize = 0;
    int numBlocks = 0;
    hipOccupancyMaxPotentialBlockSize(&numBlocks, &blockSize, vectorAdd);
    // Potential: 768, 152
    std::cout << "Potential: " << blockSize << ", " << numBlocks << std::endl;

    vectorAdd<<<numBlocksPerGrid, numThreadsInBlock, sizeof(float) * N>>>(ptr1, ptr2, N);
    // GPU kernels are asynchronous with host by default
    int numItr = 5;
    while (numItr--)
    {
        std::cout << "cpu itr: " << numItr << std::endl;
        std::this_thread::sleep_for(1s);
    }

    hipEventRecord(stop, 0);
    // Waits for an event to complete.
    hipEventSynchronize(stop);

    float elapsedTimeInMs;
    hipEventElapsedTime(&elapsedTimeInMs, start, stop);

    std::cout << "kernel runs: " << elapsedTimeInMs << " ms " << std::endl;

    res = v1Device;
    return res;
}

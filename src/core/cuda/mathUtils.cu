#include "hip/hip_runtime.h"

#include <mathUtils.h>

namespace core::cuda
{
    __device__ int pow2ceil(int n)
    {
        int pow2 = 1 << (31 - __clz(n));
        if (n > pow2)
            pow2 = (pow2 << 1);
        return pow2;
    }
}
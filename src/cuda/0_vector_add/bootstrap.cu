#include <cmath>
#include <chrono>
#include <random>
#include <format>
#include <iostream>

// Thrust: The C++ Parallel Algorithms Library
#include <thrust/host_vector.h>
#include <cuDevice.h>
#include <cuPredefines.h>
#include <kernel.h>

using namespace std::chrono;
using namespace core::cuda;

extern float run_kernel_reduce(thrust::host_vector<float> &);

int main(int argc, char **argv)
{
    checkDevice();

    thrust::host_vector<float> v1Host = {1, 2, 3, 4, 5};
    thrust::host_vector<float> v2Host = {1, 2, 3, 4, 5};

    auto res = runVectorAdd(v1Host, v2Host, 1, 5);

    CUDACHECK(hipPeekAtLastError());

    // wait for gpu to finish
    // like fence
    // device synchronization
    hipDeviceSynchronize();


    for (int i = 0; i < res.size(); i++)
    {
        std::cout << res[i] << std::endl;
    }

    return 0;
}
#include "hip/hip_runtime.h"
#include <algorithm>
#include <thrust/device_vector.h>
#include "kernel.h"

#if defined(V1)
// v2 is ~ 3 seconds in a2000
// cuda kernel function always return void, pass in pointer to write
// reduce pair per dt: pass1
// reduce pair per dt/2: pass2
__global__ void kernel_reduce_pass(float *x, int dt)
{
    int globalThreadId = blockDim.x * blockIdx.x + threadIdx.x;
    // two global memory access
    x[globalThreadId] += x[globalThreadId + dt];
}

float run_kernel_reduce(thrust::host_vector<float> &vHost)
{
    auto N = vHost.size();
    thrust::device_vector<float> vDevice(N);
    // copy data from host to device through PCI-bus
    vDevice = vHost;

    // iteration passes
    auto dt = N / 2;
    while (dt > 0)
    {
        // considering warp size
        auto numThreadsInBlock = std::min(256ul, dt);
        auto numThreadBlocks = std::max(dt / 256, 1ul);
        kernel_reduce_pass<<<numThreadBlocks, numThreadsInBlock>>>(vDevice.data().get(), dt);
        dt /= 2;
    }
    // waiting for all the device op done.
    hipDeviceSynchronize();
    // device -> host by demand, only one element
    auto result = vDevice[0];
    return result;
}

#elif defined(V2)
// v2 is less than 2 seconds in a2000
// N for boundary check
__global__ void kernel_reduce_pass(float *x, int N)
{
    int globalThreadId = blockDim.x * blockIdx.x + threadIdx.x;
    float tsum = 0.0f;
    // this stride guarenteed no threads will overlap.
    // stride: max range kernel cover.
    // if N > stride, do sum here and stored in the lowerst offset.
    int stride = gridDim.x * blockDim.x;
    for (int k = globalThreadId; k < N; k += stride)
    {
        tsum += x[k];
    }
    x[globalThreadId] = tsum;
}

float run_kernel_reduce(thrust::host_vector<float> &vHost)
{
    auto N = vHost.size();
    thrust::device_vector<float> vDevice(N);
    // copy data from host to device through PCI-bus
    vDevice = vHost;

    auto numThreadsInBlock = 256;
    auto numThreadBlocks = 128;

    kernel_reduce_pass<<<numThreadsInBlock, numThreadBlocks>>>(vDevice.data().get(), N);
    // after pass1: all the partial sum result is in range: [0, numThreadsInBlock * numThreadBlocks]
    // all the threads should be passed to 32 warp engine
    // stride: 1 * numThreadsInBlock
    // numThreadBlocks elements are partial summed in pass2
    kernel_reduce_pass<<<1, numThreadsInBlock>>>(vDevice.data().get(), numThreadsInBlock * numThreadBlocks);
    // pass3: aggregate: numThreadsInBlock
    // stride is 1
    kernel_reduce_pass<<<1, 1>>>(vDevice.data().get(), numThreadsInBlock);
    // waiting for all the device op done.
    hipDeviceSynchronize();
    // device -> host by demand, only one element
    auto result = vDevice[0];
    return result;
}

#endif

#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include "kernel.h"

// cuda kernel function always return void, pass in pointer to write
__global__ void kernel_integrate(
    float *sumArrayW, int numSteps, int numTerms, float stepSize)
{
    // threads in a warp and sm are continuous access the memory
    // gridDim.xyz: number of thread blocks in the grid
    // blockDim.xyz: number of threads in a block in three direction
    // blockIdx.x: [0, gridDim.x - 1];
    // gridDim.x * blockDim.x: all the number of threads in x direction.

    auto step = blockIdx.x * blockDim.x + threadIdx.x;

    if (step < numSteps)
    {
        int x = stepSize * step;
        sumArrayW[x] = sinAtX(x, numTerms);
    }
}

float run_kernel_integrate(int numBlocks, int numThreadsInBlock, int numSteps, int numTerms, float stepSize)
{
    // gpu vector can only be compiled in cuda
    thrust::device_vector<float> sumArrayW(numSteps);
    float *ptr = thrust::raw_pointer_cast(&sumArrayW[0]);

    // 1D grid
    // how many threads: numThreadsInBlock * numBlocks
    kernel_integrate<<<numBlocks, numThreadsInBlock>>>(
        ptr,
        numSteps,
        numTerms,
        stepSize);

    // reduce is host runnable
    auto sum = thrust::reduce(sumArrayW.begin(), sumArrayW.end());

    return sum;
}
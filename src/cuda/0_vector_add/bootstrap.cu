#include <cmath>
#include <chrono>
#include <random>
#include <format>
#include <iostream>

// Thrust: The C++ Parallel Algorithms Library
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/mr/host_memory_resource.h>
#include <thrust/mr/device_memory_resource.h>
#include <thrust/mr/universal_memory_resource.h>
#include <thrust/mr/allocator.h>

#include <cuDevice.h>
#include <cuPredefines.h>
#include <kernel.h>

using namespace std::chrono;
using namespace core::cuda;

extern float run_kernel_reduce(thrust::host_vector<float> &);

int main(int argc, char **argv)
{
    selectDevice();

    thrust::host_vector<float> v1Host = {1, 2, 3, 4, 5};
    thrust::host_vector<float> v2Host = {1, 2, 3, 4, 5};
    //   - New `thrust::universal_host_pinned_memory_resource` for allocating memory
    //   that can be accessed from the host and the device but always resides in
    //   host memory (e.g. `hipHostMalloc`).

    /*! The host pinned memory resource for the CUDA system. Uses
     *  <tt>hipHostMalloc</tt> and wraps the result with \p
     *  cuda::universal_pointer.
     */

    thrust::host_vector<int, thrust::mr::stateless_resource_allocator<int, thrust::universal_host_pinned_memory_resource>> v1Device = v1Host;
    auto raw_ptr = thrust::raw_pointer_cast(v1Device.data());



    for (int i = 0; i < v1Device.size(); i++)
    {
        std::cout << v1Device[i] << std::endl;
    }

    float scaleFactor[NUM_SCALEFACTOR] = {2.0f};
    setScaleFactor(scaleFactor);

    //   Max Threads per block 1024.0
    // exceed 1024, fail the kernel
    auto res = runVectorAdd(v1Host, v2Host, 1024);
    // auto res = runVectorAdd(v1Host, v2Host, 2048);

    // CUDACHECK(hipPeekAtLastError());

    // wait for gpu to finish
    // like fence
    // device synchronization
    hipDeviceSynchronize();

    for (int i = 0; i < res.size(); i++)
    {
        std::cout << res[i] << std::endl;
    }

    return 0;
}
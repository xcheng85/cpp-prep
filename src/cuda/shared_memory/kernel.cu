#include "hip/hip_runtime.h"
#include <algorithm>
#include <thrust/device_vector.h>
#include <mathUtils.h>
#include "kernel.h"

using namespace core::cuda;

#if defined(SHARED_MEMORY)
// output's size is numThreadBlocks
// each thread block uses shared_memory to compute 1 value and write it to the output
__global__ void kernel_reduce_pass(float *__restrict input, float *__restrict output, int N)
{
    // to use shared_memory, focus on threadIdx
    // shared_memory is per-thread block.
    // how many shared_memory resources are used ?
    // 48K per thread block: 48 * 1024 > 256 * 4 byte
    // numThreads * float

    // dynamic shared memory allocation, size is provided at 3rd param in the kernal launch
    // static shared memory allocation ex: __shared__ float partialsum[256];
    extern __shared__ float partialsum[];

    int globalThreadId = blockDim.x * blockIdx.x + threadIdx.x;
    int localThreadId = threadIdx.x;
    // stride to make unique access to global memory
    int stride = gridDim.x * blockDim.x;
    partialsum[localThreadId] = 0.f;
    for (int k = globalThreadId; k < N; k += stride)
    {
        partialsum[localThreadId] += input[k];
    }
    // partialsum[] are all filled
    // https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html
    __syncthreads();

    // partialsum[localThreadId] are ready to reduce: assume 256 threads in a block
    // [lowhalf, highhalf]
    // [lowhalf + highhalf, highhalf]: lowhalf is updated

    // if blockDim is not power of 2
    // 40 SM
    // Warp-size: 32
    // each SM: pow(2, m) * warp-size
    // Max Threads per Block: 1024
    // SM: m * warp

    auto dt = pow2ceil(blockDim.x) / 2;

    while (dt > 0)
    {
        // extra guard due to ceil op
        if (localThreadId < dt && localThreadId + dt < blockDim.x)
        {
            partialsum[localThreadId] += partialsum[localThreadId + dt];
        }
        // ensure all write is done, need to use updated value in the next while-iteration
        __syncthreads();
        dt /= 2;
    }

    // partialsum[0] is the sum of all this thread block
    if (localThreadId == 0)
    {
        output[blockIdx.x] = partialsum[0];
    }
}

#elif defined(WARP_OP)
// output's size is numThreadBlocks
// each thread block uses shared_memory to compute 1 value and write it to the output
__global__ void kernel_reduce_pass(float *__restrict input, float *__restrict output, int N)
{
    // to use shared_memory, focus on threadIdx
    // shared_memory is per-thread block.
    // how many shared_memory resources are used ?
    // 48K per thread block: 48 * 1024 > 256 * 4 byte
    // numThreads * float

    extern __shared__ float partialsum[];

    int globalThreadId = blockDim.x * blockIdx.x + threadIdx.x;
    int localThreadId = threadIdx.x;
    // stride to make unique access to global memory
    int stride = gridDim.x * blockDim.x;
    partialsum[localThreadId] = 0.f;
    for (int k = globalThreadId; k < N; k += stride)
    {
        partialsum[localThreadId] += input[k];
    }
    // partialsum[] are all filled
    // https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html
    __syncthreads();

    // partialsum[localThreadId] are ready to reduce: assume 256 threads in a block
    // [lowhalf, highhalf]
    // [lowhalf + highhalf, highhalf]: lowhalf is updated

    // if blockDim is not power of 2
    // 40 SM
    // Warp-size: 32
    // each SM: pow(2, m) * warp-size
    // Max Threads per Block: 1024
    // SM: m * warp

    // dt: 256, 128, ..., 0
    // assuming blockDim is not too big
    auto dt = pow2ceil(blockDim.x) / 2;
    // loop unrolling optimization
    // the if condition protected the read after write race condition.
    
    if (localThreadId < 256 && localThreadId + 256 < blockDim.x)
    {
        // no read-write race condition here,
        // no one is writing to partialsum[localThreadId + 256] at this stage
        partialsum[localThreadId] += partialsum[localThreadId + 256];
    }
    __syncthreads();
    if (localThreadId < 128)
    {
        // no read-write race condition here,
        partialsum[localThreadId] += partialsum[localThreadId + 128];
    }
    __syncthreads();
    if (localThreadId < 64)
    {
        // no read-write race condition here,
        partialsum[localThreadId] += partialsum[localThreadId + 64];
    }
    __syncthreads();
    if (localThreadId < 32)
    {
        // no read-write race condition here,
        partialsum[localThreadId] += partialsum[localThreadId + 32];
    }
    __syncthreads();

    // access is within warp-size: 32
    if (localThreadId < 16)
    {
        // no read-write race condition here,
        partialsum[localThreadId] += partialsum[localThreadId + 16];
    }
    __syncwarp();
    if (localThreadId < 8)
    {
        // no read-write race condition here,
        partialsum[localThreadId] += partialsum[localThreadId + 8];
    }
    __syncwarp();
    if (localThreadId < 4)
    {
        // no read-write race condition here,
        partialsum[localThreadId] += partialsum[localThreadId + 4];
    }
    __syncwarp();
    if (localThreadId < 2)
    {
        // no read-write race condition here,
        partialsum[localThreadId] += partialsum[localThreadId + 2];
    }
    __syncwarp();

    if (localThreadId == 0)
    {
        output[blockIdx.x] = partialsum[0] + partialsum[1];
    }
}

#endif

float run_kernel_reduce(thrust::host_vector<float> &vHost)
{
    auto N = vHost.size();
    thrust::device_vector<float> vDeviceInput(N);
    // copy data from host to device through PCI-bus
    vDeviceInput = vHost;
    auto numThreadsInBlock = 256;
    auto numThreadBlocks = 128;
    // each thread block will write to one value to this array
    thrust::device_vector<float> vDeviceOutput(numThreadBlocks);

    // third parameter: shared_memory size in bytes
    auto sharedMemorySizeInBytes = numThreadsInBlock * sizeof(float);
    kernel_reduce_pass<<<numThreadBlocks, numThreadsInBlock, sharedMemorySizeInBytes>>>(
        vDeviceInput.data().get(),
        vDeviceOutput.data().get(),
        N);
    // pass 2:
    // ping-pong, output of pass1 became input of pass2.
    // input's size: numThreadBlocks of pass1
    sharedMemorySizeInBytes = numThreadBlocks * sizeof(float);
    kernel_reduce_pass<<<1, numThreadBlocks, sharedMemorySizeInBytes>>>(
        vDeviceOutput.data().get(),
        vDeviceInput.data().get(),
        numThreadBlocks);

    // waiting for all the device op done.
    hipDeviceSynchronize();
    // device -> host by demand, only one element
    auto result = vDeviceInput[0];
    return result;
}
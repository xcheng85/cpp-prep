#include "hip/hip_runtime.h"
#include <algorithm>
#include <assert.h>
#include <thrust/device_vector.h>
#include <cuPredefines.h>
#include "kernel.h"

// v1: to be overwritten
__global__ void vectorAdd(float *v1, float *v2, int N)
{
    int globalThreadId = blockDim.x * blockIdx.x + threadIdx.x;
    // two global memory access
    // debug in kernel
    if (globalThreadId < N)
    {
        printf("calling kernel: %d\n", globalThreadId);
        v1[globalThreadId] += v2[globalThreadId];
    }
}

thrust::host_vector<float> runVectorAdd(thrust::host_vector<float> &v1,
                                        thrust::host_vector<float> &v2,
                                        int numThreadsInBlock)
{
    thrust::host_vector<float> res;
    assert(v1.size() == v2.size());
    auto N = v1.size();
    thrust::device_vector<float> v1Device(N), v2Device(N);

    // copy data from host to device through PCI-bus
    // deep down using hipMemcpy

    v1Device = v1;
    v2Device = v2;

    float *ptr1 = thrust::raw_pointer_cast(&v1Device[0]);
    float *ptr2 = thrust::raw_pointer_cast(&v2Device[0]);

    hipEvent_t start, stop;
    CUDACHECK(hipEventCreate(&start));
    CUDACHECK(hipEventCreate(&stop));

    // stream 0
    hipEventRecord(start, 0);
    int numBlocksPerGrid = (N + numThreadsInBlock - 1) / numThreadsInBlock;
    vectorAdd<<<numBlocksPerGrid, numThreadsInBlock>>>(ptr1, ptr2, N);
    hipEventRecord(stop, 0);
    // Waits for an event to complete.
    hipEventSynchronize(stop);

    float elapsedTimeInMs;
    hipEventElapsedTime(&elapsedTimeInMs, start, stop);

    std::cout << "kernel runs: " << elapsedTimeInMs << " ms " << std::endl;

    res = v1Device;
    return res;
}

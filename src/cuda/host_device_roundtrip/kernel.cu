#include "hip/hip_runtime.h"
#include <algorithm>
#include <thrust/device_vector.h>
#include "kernel.h"

// cuda kernel function always return void, pass in pointer to write
// reduce pair per dt: pass1
// reduce pair per dt/2: pass2
__global__ void kernel_reduce_pass(float *x, int dt)
{
    int globalThreadId = blockDim.x * blockIdx.x + threadIdx.x;
    x[globalThreadId] += x[globalThreadId + dt];
}

float run_kernel_reduce_v1(thrust::host_vector<float> &vHost)
{
    auto N = vHost.size();
    thrust::device_vector<float> vDevice(N);
    // copy data from host to device through PCI-bus
    vDevice = vHost;

    // iteration passes
    auto dt = N / 2;
    while (dt > 0)
    {
        // considering warp size
        auto numThreadsInBlock = std::min(256ul, dt);
        auto numThreadBlocks = std::max(dt / 256, 1ul);
        kernel_reduce_pass<<<numThreadBlocks, numThreadsInBlock>>>(vDevice.data().get(), dt);
        dt /= 2;
    }
    // waiting for all the device op done.
    hipDeviceSynchronize();
    // device -> host by demand, only one element
    auto result = vDevice[0];
    return result;
}
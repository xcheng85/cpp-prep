#include "hip/hip_runtime.h"
#include <concepts>
#include <hip/hip_cooperative_groups.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <kernel.h>

using namespace cooperative_groups;

// each launched kernel (1D case) has number of threads in total
// gridDim.x * blockDim.x

// assuming input size >> number of threads launched in kernel
// pass1: reduce everything beyond (gridDim.x * blockDim.x) to the (gridDim.x * blockDim.x)
// pass2: reduce everything beyond (blockDim.x) to the result of thread group0.
__device__ float reducePass1(float *__restrict input, int n)
{
    float sum = 0;

    // vector memory access
    for (int i = blockIdx.x * blockDim.x + threadIdx.x;
         i < n / 4;
         i += blockDim.x * gridDim.x)
    {
        float4 in = ((float4 *)input)[i];
        sum += in.x + in.y + in.z + in.w;
    }
    return sum;
}

// tg is provided runtime
__device__ float reducePass2(thread_group tg, float *__restrict outputSM, float reduceResultFromPass1PerThread)
{
    // goal in pass2, is reduce everything to the tg granulariy
    int threadIdInTg = tg.thread_rank();
    int threadGroupSize = tg.size();

    for (int dt = threadGroupSize / 2; dt > 0; dt /= 2)
    {
        // shared memory is isolated per tg.
        outputSM[threadIdInTg] = reduceResultFromPass1PerThread;
        // cannot use warpsync here due to size could > 32.
        // ensure all the output[threadIdInTg] is written for all threads
        tg.sync();

        if (threadIdInTg < dt)
        {
            // is a copy anyway
            reduceResultFromPass1PerThread += outputSM[threadIdInTg + dt];
        }
        // make sure read of outputSM is done before overwritten in the next for iteration
        tg.sync();
    }
    return reduceResultFromPass1PerThread;
}

// template-meta programming: compiler optimzation
template <typename TG>
concept IsThreadGroup = requires(TG x) {
    x.thread_rank();
    x.size();
};

// template <std::integral T>
// constexpr bool isWarpSize = (T == 32);

// integral concept
// created from the type_traits value
// template< class T >
// concept integral = std::is_integral_v<T>;
template <typename T>
concept IsWarpSizeThreadGroup = std::integral<T>;
//&& isWarpSize<T>;

template <int TG>
__device__ float reduceWithinThreadGroup(thread_block_tile<TG> tg, float reduceResultFromPass1PerThread)
{
    // goal in pass2, is reduce everything to the tg granulariy
    int threadGroupSize = tg.size();

// threadGroupSize is known in compile time
// with shfl_down, no need to use shared memory
#pragma unroll
    for (int dt = threadGroupSize / 2; dt > 0; dt /= 2)
    {
        // Now if the TG is in warp-size granularity
        // which is guarded by the c++ concept
        // https://developer.nvidia.com/blog/faster-parallel-reductions-kepler/
        reduceResultFromPass1PerThread += tg.shfl_down(reduceResultFromPass1PerThread, dt);
    }
    return reduceResultFromPass1PerThread;
}

#if defined(TILED_PARTITION)
// indent is needed
#if defined(TEMPLATE_META)
template <int TileSize>
__global__ void kernel_reduce(float *__restrict input, float *__restrict output, int n)
{
    float sumGlobalThread = reducePass1(input, n);
    // cannot do this + template
    // thread_group tg = this_thread_block();
    thread_block_tile<TileSize> tile = tiled_partition<TileSize>(this_thread_block());
    int sumTile = reduceWithinThreadGroup<TileSize>(tile, sumGlobalThread);

    if (tile.thread_rank() == 0)
    {
        // number of atomicAdd will more after the tiled_partition
        // every warp-size
        printf("[template metaprogramming]: atomicAdd: %d:\n", blockIdx.x * blockDim.x + threadIdx.x);
        atomicAdd(output, sumTile);
    }
}
#else
__global__ void kernel_reduce(float *__restrict input, float *__restrict output, int n)
{
    float sumGlobalThread = reducePass1(input, n);

    extern __shared__ float sharedmemory[];

    thread_group tg = this_thread_block();

    // divide within thread block by warp-size
    auto tileIdx = tg.thread_rank() / 32;
    // share memory size remains unchanged.
    // but reference with offset
    float *sharedmemory32 = &sharedmemory[32 * tileIdx];

    // tiled_partition return same type: thread_group, just size is 32, known compile time
    thread_group tg32 = tiled_partition(tg, 32);

    // for each threadBlock, only the first thread in that block has the final sum per block
    float sumBlockThread = reducePass2(tg32, sharedmemory32, sumGlobalThread);

    if (tg32.thread_rank() == 0)
    {
        // number of atomicAdd will more after the tiled_partition
        // every warp-size
        printf("atomicAdd: %d:\n", blockIdx.x * blockDim.x + threadIdx.x);
        atomicAdd(output, sumBlockThread);
    }
}
#endif
#else
__global__ void kernel_reduce(float *__restrict input, float *__restrict output, int n)
{
    float sumGlobalThread = reducePass1(input, n);

    extern __shared__ float sharedmemory[];

    thread_group tg = this_thread_block();

    // for each threadBlock, only the first thread in that block has the final sum per block
    float sumBlockThread = reducePass2(tg, sharedmemory, sumGlobalThread);

    if (tg.thread_rank() == 0)
    {
        // number of printf should = numThreadBlocks;
        printf("atomicAdd: %d:\n", blockIdx.x * blockDim.x + threadIdx.x);
        atomicAdd(output, sumBlockThread);
    }
}
#endif

float run_kernel(thrust::host_vector<float> &vHost)
{
    auto N = vHost.size();
    thrust::device_vector<float> vDeviceInput(N);
    // copy data from host to device through PCI-bus
    vDeviceInput = vHost;
    auto numThreadsInBlock = 64;
    auto numThreadBlocks = 4;

    thrust::device_vector<float> vDeviceOutput(1);
    auto sharedMemorySizeInBytes = numThreadsInBlock * sizeof(float);

#ifdef TEMPLATE_META
    kernel_reduce<32><<<numThreadBlocks, numThreadsInBlock>>>(
        vDeviceInput.data().get(),
        vDeviceOutput.data().get(),
        N);
#else
    kernel_reduce<<<numThreadBlocks, numThreadsInBlock, sharedMemorySizeInBytes>>>(
        vDeviceInput.data().get(),
        vDeviceOutput.data().get(),
        N);
#endif

    // gpu->cpu
    return vDeviceOutput[0];
}